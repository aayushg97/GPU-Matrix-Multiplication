#include "hip/hip_runtime.h"
// ;-*- mode: c;-*-
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include "../src/utils.h"
#include "../src/types.h"
#include "mytypes.h"
using namespace std;

#include <stdio.h>

#ifdef NAIVE
__global__ void matMul(int N, _FTYPE_ *C, _FTYPE_ *A, _FTYPE_ *B) {

    int I =  blockIdx.y*blockDim.y + threadIdx.y;
    int J =  blockIdx.x*blockDim.x + threadIdx.x;

    if((I < N) && (J < N)){
        _FTYPE_ _c = 0;
        for (unsigned int k = 0; k < N; k++) {
            _FTYPE_ a = A[I * N + k];
            _FTYPE_ b = B[k * N + J];
            _c += a * b;
        }
        C[I * N + J] = _c;
    }
}

#else
extern __shared__ _FTYPE_ sharmem[];

//You should be changing the kernel here for the non naive implementation.
__global__ void matMul(int N, _FTYPE_ * __restrict__ C, _FTYPE_ * __restrict__ A, _FTYPE_ * __restrict__ B) {
	//__shared__ _FTYPE_ As[TILEDIM_M][TILEDIM_K], Bs[TILEDIM_K][TILEDIM_N];
	_FTYPE_ * __restrict__ As = &sharmem[0];
	_FTYPE_ * __restrict__ Bs = &As[TILEDIM_M * TILEDIM_K];
	
	int ty = threadIdx.y, tx = threadIdx.x;
	int by = blockIdx.y, bx = blockIdx.x;
	
    	int I =  blockIdx.y*TILEDIM_M + threadIdx.y;
    	int J =  blockIdx.x*TILEDIM_N + threadIdx.x;
	
	register _FTYPE_ Cij[TILESCALE_M][TILESCALE_N] = {0};
	
	int kk = 0;
	int numTiles = N/TILEDIM_K;
	if(N % TILEDIM_K != 0)
		numTiles++;

	#pragma unroll
	for(kk=0; kk<numTiles; kk++)
	{
		// Assuming that TILEDIM is same in m, n directions
		#pragma unroll
		for(int row=0; row<TILEDIM_M; row+=TILESTEP_M)
		{
			#pragma unroll
			for(int col=0; col<TILEDIM_K; col+=TILESTEP_K)
			{
				As[(ty+row)*TILEDIM_K + tx + col] = A[(I+row)*N + kk*TILEDIM_K + tx + col];
			}
		}

		#pragma unroll
		for(int row=0; row<TILEDIM_K; row+=TILESTEP_K)
		{
			#pragma unroll
			for(int col=0; col<TILEDIM_N; col+=TILESTEP_N)
			{
				Bs[(ty+row)*TILEDIM_N + tx + col] = B[(kk*TILEDIM_K+ty+row)*N + J + col];
			}
		}
		
		__syncthreads();
		
		#pragma unroll
		for(int k=0; k<TILEDIM_K; k++)
		{
			#pragma unroll
			for(int row=0; row<TILEDIM_M; row+=TILESTEP_M)
			{
				#pragma unroll
				for(int col=0; col<TILEDIM_N; col+=TILESTEP_N)
				{
						Cij[row/TILESTEP_M][col/TILESTEP_N] += As[(ty+row)*TILEDIM_K + k]*Bs[k*TILEDIM_N + tx + col];
				}
			}
		}
	
		__syncthreads();
	}
	
	#pragma unroll
	for(int row=0; row<TILEDIM_M; row+=TILESTEP_M)
	{
		#pragma unroll
		for(int col=0; col<TILEDIM_N; col+=TILESTEP_N)
		{
			if(((I+row)<N)&&((J+col)<N)){
				C[(I+row)*N + J + col] = Cij[row/TILESTEP_M][col/TILESTEP_N];
			}
		}
	}	
}
#endif
